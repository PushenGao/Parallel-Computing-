#include "hip/hip_runtime.h"

#include <stdio.h>
#include <time.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define Simple	0
#define Multi	1

#define Critical 1e-2
#define M_Critical 1e-3
#define debug
#define MAX_QUEENS 500		

int n ;
int goal ;					
clock_t start, end ;		
time_t	start_time, end_time ;

// queen stores the DNA of a individual 
// unitFitness is the fitness of the individual
// eachFitness is every gene’s fitness
typedef struct {
	int queen[MAX_QUEENS] ;
	int unitFitness	;
	int eachFitness[MAX_QUEENS] ;
} Population ; 

Population s_population, m_population[300] ;
int m_size ;
int m_totFitness ;    

void init ()
{
	srand (0) ;
	scanf ("%d", &n) ;
	goal = n * (n - 1) ;
	m_size = 300 ;
}

int compare(const void *a,const void *b)
{
  return ((Population *)b)->unitFitness - ((Population *)a)->unitFitness;
}  

int Aggressive(Population *p, int i, int j)
{
	return (abs(p->queen[i] - p->queen[j]) == abs(i - j)) ? 0 : 1 ;
}

void UpdateFitnessScore (Population *p) 
{
	int i, j;

	p->unitFitness = 0 ;
	for (i = 0 ; i < n ; i++)
	{
		p->eachFitness[i] = 0 ;
		for (j = 0 ; j < n ; j++)
			p->eachFitness[i] += Aggressive(p, i, j) ;

		p->unitFitness += p->eachFitness[i] ;     
	}

}


void CreateMultiStartPopulation ()
{	
	int loop, i, j ;
	int tmp[MAX_QUEENS] ;
	
	for (loop = 0 ; loop < m_size ; loop ++)	
	{
		for (i = 0 ; i < n ; i++)
				tmp[i] = i ;

			for (i = 0 ; i < n ; i++)
			{
				j = rand() % (n - i) ;
				m_population[loop].queen[i] = tmp[j] ;
				tmp[j] = tmp[n - i - 1] ;
			}
			UpdateFitnessScore(&m_population[loop]) ;
	}
}


void MultiMutate (Population* p)
{
	int i, j, swap ;
	int worst ;
	Population baby ;

	worst = 0 ;
	for (i = 0 ; i < n ; i++)
		if (p->eachFitness[i] < p->eachFitness[worst])
			worst = i ;

	baby = *p ;
	for (i = 0 ; i < n / 4 ; i++)
	{
		j = rand() % n ;	
			
		swap = baby.queen[worst] ;
		baby.queen[worst] = baby.queen[j] ;
		baby.queen[j] = swap ;

		UpdateFitnessScore(&baby) ;
		if (baby.unitFitness > p->unitFitness || (double)rand() / RAND_MAX < M_Critical)
		{
			*p = baby ;
			break ;
		}
	}
}

int RouletteWheelSelection()
{
	int selection = 0;
	int i ;

	double slice = (double)rand() / RAND_MAX;
	double addFitness = 0;
	for(i = 0; i < m_size ; i++)
	{
		addFitness +=  (double)m_population[i].unitFitness / m_totFitness ;
		if(addFitness > slice)
		{
			selection = i;
			break;
		}

	}
	return selection;
}


//calculate the crossover function in GPU with 256 threads and 1 block
__global__ static void cudaCross(int *pQueen, 
				 //int *pUnitFitness, 
				 //int *pEachFitness,
				 int *cQueen, 
				 int *cUnitFitness, 
				 int *cEachFitness,
				 int *mateSeq,
				 int *positionSeq,
				 int n,
				 int m_size,
				 int THREAD_NUM){
	int tid = threadIdx.x;
	int count;
	for (count = tid ; count < m_size - 2 ; count=count+THREAD_NUM)
	{
		int flag[500] ;
		int pos1, pos2, tmp, father, mother ;
		int i, j ;
		
	
		pos1 = positionSeq[count];
		pos2 = positionSeq[count+1];
		if (pos1 > pos2) { tmp = pos1 ; pos1 = pos2 ; pos2 = tmp; }
		
		father = mateSeq[(count)*2];
		mother = mateSeq[(count)*2+1];
		for (j = 0 ; j < n ; j ++)
			flag[j] = 0 ;
		for (j = pos1; j <= pos2; j++)
			flag[pQueen[father*n+j]] = 1 ;

		for(i = 0, j = 0 ; i < n ; i++)
		{
			if (i < pos1 || i > pos2) {
				while (flag[pQueen[mother*n+j]]) j++ ;
				cQueen[count*n + i] = pQueen[mother*n+j] ;
				j ++ ;
			}
			else cQueen[count*n + i] = pQueen[father*n+j] ;
		}

		cUnitFitness[count] = 0 ;
		for (i = 0 ; i < n ; i++)
		{
			cEachFitness[count*n + i] = 0 ;
			for (j = 0 ; j < n ; j++)
				cEachFitness[count*n + i] += (abs(cQueen[count*n + i] - cQueen[count*n + j]) == abs(i - j)) ? 0 : 1 ;

			cUnitFitness[count] += cEachFitness[count*n + i] ;     
		}
		//UpdateFitnessScore (baby) ;
		//CrossOverFM (m_population[father], m_population[mother], &p[count]) ; 
	}
}
//the crossover function is calculated on the GPU
void CrossOver () 
{
	int i,j,pos1,pos2;
	int h_parentSeq[m_size*2];
	int h_positionSeq[m_size*2];
	int h_pQueen[m_size*n];
	int h_cQueen[m_size*n];
	int h_cUnitFitness[m_size];
	int h_cEachFitness[m_size*n];

	int d_parentSeq[m_size*2];
	int d_positionSeq[m_size*2];
	int d_pQueen[m_size*n];
	int d_cQueen[m_size*n];
	int d_cUnitFitness[m_size];
	int d_cEachFitness[m_size*n];

	
	m_totFitness = 0 ;
	for (i = 0 ; i < m_size ; i++)
		m_totFitness += m_population[i].unitFitness ;

	for (i = 0 ; i < m_size*2 ; i=i+2){
		h_parentSeq[i] = RouletteWheelSelection () ;
		h_parentSeq[i+1] = RouletteWheelSelection () ;
		do {
		pos1 = rand() % n ;
		pos2 = rand() % n ;
		} while (pos1 == pos2) ;
		h_positionSeq[i] = pos1;
		h_positionSeq[i+1] = pos2;

	}

	for (i = 0 ; i < m_size ; i++)
		for (j = 0 ; j < n ; j++)
			h_pQueen[i*n+j] = m_population[i].queen[j];

	
	// this place should be the cuda crossover
	/***************************************************
	  1st Part: Allocation of memory on device memory  
	    ****************************************************/	    
	    
	    hipMalloc((void**) &d_parentSeq, sizeof(int) * m_size * 2);
	    hipMalloc((void**) &d_positionSeq, sizeof(int) * m_size * 2);
	    hipMalloc((void**) &d_pQueen, sizeof(int) * m_size * n);
	    hipMalloc((void**) &d_cQueen, sizeof(int) * m_size * n);
	    hipMalloc((void**) &d_cUnitFitness, sizeof(int) * m_size);
	    hipMalloc((void**) &d_cEachFitness, sizeof(int) * m_size * n);
	    //How to write Memcpy
	    hipMemcpy(d_parentSeq, h_parentSeq, sizeof(int) * m_size * 2, hipMemcpyHostToDevice);
	    hipMemcpy(d_pQueen, h_pQueen, sizeof(int) * m_size * n, hipMemcpyHostToDevice);
	    hipMemcpy(d_positionSeq, h_positionSeq, sizeof(int) * m_size * 2, hipMemcpyHostToDevice);
	    //hipMemcpy(sq_matrix_2_d, sq_matrix_2, size, hipMemcpyHostToDevice);   
	 
	    
	    /***************************************************
	   2nd Part: Inovke kernel 
	    ****************************************************/
	    int THREAD_NUM = 256;
	    cudaCross<<<1, THREAD_NUM, 0>>>(d_pQueen, 
					 //int *pUnitFitness, 
					 //int *pEachFitness,
					 d_cQueen, 
					 d_cUnitFitness, 
					 d_cEachFitness,
					 d_parentSeq,
					 d_positionSeq,
					 n,
					 m_size,
					 THREAD_NUM);
	    
	    /***************************************************
	   3rd Part: Transfer result from device to host 
	    ****************************************************/
	    hipMemcpy(h_cQueen, d_cQueen, sizeof(int) * m_size * n, hipMemcpyDeviceToHost);
	    hipMemcpy(h_cUnitFitness, d_cUnitFitness, sizeof(int) * m_size, hipMemcpyDeviceToHost);
	    hipMemcpy(h_cEachFitness, d_cEachFitness, sizeof(int) * m_size * n, hipMemcpyDeviceToHost);

	    hipFree(d_parentSeq);
	    hipFree(d_pQueen);
	    hipFree(d_cQueen);
	    hipFree(d_cUnitFitness);
	    hipFree(d_cEachFitness);
	for (int count = 0 ; count < m_size - 2 ; count++){
		m_population[count+2].unitFitness = h_cUnitFitness[count] ;
		for (j = 0; j < n ; j++){
			m_population[count+2].queen[j] = h_cQueen[count*n+j];
			m_population[count+2].eachFitness[j] = h_cEachFitness[count*n+j];
		}
	}
}

void PrintQueens (Population p)
{
	double secs ;

	secs = (double)(end - start) / (CLOCKS_PER_SEC*20*8) ;
    printf("Calculations took %.3lf second%s.\n", secs, (secs < 1 ? "" : "s"));
	

}

int main ()
{
	
	/*
	freopen ("input.txt", "r", stdin) ;
	freopen ("output.txt", "w", stdout) ;
	*/

	while (1) 
	{
		
		init() ;	
		if (n == 0) break ;
		
		
		
		
		/*
		start = clock () ;
		time (&start_time) ;	
		printf("With Single Population : \nStart: \t %s", ctime(&start_time));

		CreateSimpleStartPopulation() ;
	
		do {
			SimpleMutate () ;
		} while (s_population.unitFitness < goal) ;

		end = clock () ;
		time (&end_time) ;
		PrintQueens(s_population) ;
		*/

	
	
		time (&start_time) ;	
		printf("With Multi Population : \nStart: \t %s", ctime(&start_time));

		CreateMultiStartPopulation() ;
		
		int iterationCount=0;

		for(int i=0;i<30;i++){
			start = clock () ;
			while (iterationCount<20){
				qsort(m_population, m_size, sizeof(Population), compare) ;
				MultiMutate (&m_population[0]) ;
				MultiMutate (&m_population[1]) ;
				if (m_population[0].unitFitness == goal || m_population[1].unitFitness ==  goal)
					break ;
				CrossOver () ;
				iterationCount++;
			} 
			end = clock () ;
			PrintQueens(m_population[0].unitFitness == goal ? m_population[0] : m_population[1]) ;
			iterationCount = 0;
		}

		time (&end_time) ;
  }
}